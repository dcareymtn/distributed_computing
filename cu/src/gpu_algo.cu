#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <time.h>

#include "gpu_algo.h"

namespace gpu
{

__global__ void print_cuda(char *a, int N)
{
    char p[11]="Hello CUDA";
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    
	if(idx < N) 
    {
        a[idx]=p[idx];
    }
}

__global__ void gpu_count_occurrences_global( double *dM, int size, int start_index, int stop_index, int *d_counter )
{
	int tid = blockIdx.x * blockDim.x + threadIdx.x;
	
	if (tid > size) return;

	int bin = dM[tid] - start_index;

	atomicAdd( &d_counter[bin], 1);

}

__global__ void gpu_count_occurrences_shared( double * pM, int size, int start_index, int stop_index, int *d_counter )
{

	extern __shared__ int s_counter[];

	int tid 	= blockIdx.x * blockDim.x + threadIdx.x;
	int N = stop_index - start_index + 1;

	if (tid < (stop_index - start_index + 1))
	{	
		s_counter[tid] = 0;
		d_counter[tid] = 0;
	}

	__syncthreads();
	
	if (tid < size)
	{
		int sbin = pM[tid] - start_index;
		atomicAdd( &s_counter[sbin], 1);
	}

	__syncthreads();

	if ((threadIdx.x < N) && (tid < size))
	{
		atomicAdd( &d_counter[threadIdx.x], s_counter[threadIdx.x] );
	}

}

__global__ void gpu_rms_filter_shared( double * _d_M, int filtNRows, int filtNCols, double * _d_MResult )
{
	
	extern __shared__ int _l_M[];
	
	int subMatIdx = blockIdx.x;
	
	int blockSize	 = blockDim.x * blockDim.y;

	int row = threadIdx.x;
	int col = threadIdx.y;

	_l_M[subMatIdx * blockSize + row * blockDim.y + col] = *(_d_M + subMatIdx * blockSize + row * blockDim.y + col );

	__syncthreads();

	double this_result(0);

	int start_row 	= max(row - (filtNRows - 1)/2, 0);
	int stop_row 	= min(row + (filtNRows - 1)/2, blockDim.x);
	
	int start_col 	= max(col - (filtNCols - 1)/2, 0);
	int stop_col 	= min(col + (filtNCols - 1)/2, blockDim.y);

	double temp(0);

	for (int iRow = start_row; iRow <= stop_row; iRow++)
	{
		for (int iCol = start_col; iCol <= stop_col; iCol++)
		{

			temp 		= _l_M[ subMatIdx * blockSize + iRow * blockDim.y + iCol ];
			this_result += temp*temp;

		}	
	}
	
	*(_d_MResult + subMatIdx * blockSize + row * blockDim.y + col ) = sqrt( this_result );
}

__global__ void gpu_rms_filter_global( double * _d_M, int filtNRows, int filtNCols, double * _d_MResult )
{
	int subMatIdx = blockIdx.x;
	
	int blockSize	 = blockDim.x * blockDim.y;

	int row = threadIdx.x;
	int col = threadIdx.y;

	double this_result(0);

	int start_row 	= max(row - (filtNRows - 1)/2, 0);
	int stop_row 	= min(row + (filtNRows - 1)/2, blockDim.x);
	
	int start_col 	= max(col - (filtNCols - 1)/2, 0);
	int stop_col 	= min(col + (filtNCols - 1)/2, blockDim.y);

	double temp(0);

	for (int iRow = start_row; iRow <= stop_row; iRow++)
	{
		for (int iCol = start_col; iCol <= stop_col; iCol++)
		{
			temp 		= *(_d_M + subMatIdx * blockSize + iRow * blockDim.y + iCol );
			this_result +=  temp*temp;
		}	
	}
	
	*(_d_MResult + subMatIdx * blockSize + row * blockDim.y + col ) =  sqrt( this_result );
}

void cuda_init()
{
	double *dM;
	hipMalloc( (void **) &dM, 10*sizeof(double));
	hipFree(dM);
}

void count_occurrences( double *h_M, int nRows, int nCols, int start_count, int stop_count, bool bGlobal )
{
	
	// Copy the matrix data to the gpu
	double *d_M;
	int *d_counter;
		
	int nbins 	= stop_count - start_count + 1;

	size_t counter_size = nbins * sizeof( int );
	int *h_counter = (int *) malloc( counter_size );

	for (int ii = 0; ii < nbins; ii++)
	{
		h_counter[ii] = 0;
	}

	int N = nRows * nCols;

	size_t size = N * sizeof( double );

	hipMalloc((void **)&d_M, size);
	hipMalloc((void **)&d_counter, counter_size);

	hipMemcpy( d_M, h_M, size, hipMemcpyHostToDevice);
	
	int factor = 4;

	int blockSize = (factor * nCols < 1000 ? factor * nCols : 1000);
	int nBlock = N / blockSize + (N%blockSize == 0 ? 0 : 1);

	if (bGlobal)
	{
		gpu_count_occurrences_global<<< nBlock, blockSize  >>>(d_M, size, start_count, stop_count, d_counter );
	}
	else
	{
		gpu_count_occurrences_shared<<< nBlock, blockSize, nbins*sizeof(int)  >>>(d_M, size, start_count, stop_count, d_counter );
	}


	hipMemcpy( h_counter, d_counter, counter_size, hipMemcpyDeviceToHost);

	//for (int ii = start_count; ii <= stop_count; ii++)
	//{
	//	printf("%4d | ", ii);
	//}
	//printf("\n");
	//for (int ii = 0; ii < nbins; ii++)
	//{
	//	printf("%3d |  ", h_counter[ii]);
	//}
	//printf("\n");

	free(h_counter);
	hipFree(d_M);
	hipFree(d_counter);
}	

void rms_filter( double *hBlockMResult, double *_h_BlockM, int nRowBreak, int nColBreak, int subMatNumRows, int subMatNumCols, int nFiltRows, int nFiltCols, bool bGlobal )
{
	double *_d_BlockM, *_d_BlockMResult, *_d_BlockMResultShared;

	size_t __blockSize = nRowBreak * nColBreak * subMatNumRows * subMatNumCols * sizeof(double);

	hipMalloc((void **)&_d_BlockM, __blockSize );
	hipMalloc((void **)&_d_BlockMResult, __blockSize );
	hipMalloc((void **)&_d_BlockMResultShared, __blockSize );

	hipMemcpy( _d_BlockM, _h_BlockM, __blockSize, hipMemcpyHostToDevice );

	dim3 threadsPerBlock(subMatNumRows, subMatNumCols );

	if (bGlobal)
	{
		gpu_rms_filter_global<<< nRowBreak * nColBreak, threadsPerBlock >>>( _d_BlockM, nFiltRows, nFiltCols, _d_BlockMResult );
	}
	else
	{
		gpu_rms_filter_shared<<< nRowBreak * nColBreak, threadsPerBlock, __blockSize  >>>( _d_BlockM, nFiltRows, nFiltCols, _d_BlockMResult );	
	}

	hipMemcpy( hBlockMResult, _d_BlockMResult, __blockSize, hipMemcpyDeviceToHost );
	
	hipFree(_d_BlockM);
	hipFree(_d_BlockMResult);
	hipFree(_d_BlockMResultShared);	
}

void hello_cuda(void)
{
    char *a_h, *a_d; // _h for the host and _d for the device based pointers
    const int N = 11;
    size_t size = N * sizeof(char);

    a_h = (char *) malloc(size); // allocating the array on the host
    hipMalloc((void **) &a_d, size); // allocating the array on the device
    
    // initialize the host array
    for (int i = 0; i < N; i++)
    {
        a_h[i] = 0;
    }

    // Copy the array on the host to the device
    hipMemcpy(a_d, a_h, size, hipMemcpyHostToDevice);

    int blocksize = 4;
    int nblock = N/blocksize + (N%blocksize == 0 ? 0 : 1);
	nblock = 100;

    print_cuda <<< nblock, blocksize >>>(a_d, N); // Run the kernel on the device

	hipError_t err = hipGetLastError();
	if (err != hipSuccess)
		printf("Error: %s\n", hipGetErrorString(err));
    // copy from the device memory to the host memory
    hipMemcpy(a_h, a_d, sizeof(char) * N, hipMemcpyDeviceToHost);


	// print the array on the host
    for (int i = 0; i < N; i++)
    {
        printf("%c", a_h[i]);
    }
	printf("\n");
    free(a_h);
	hipFree(a_d);
}


void particle_swarm_eval( double (*f)(int dim, double * vec), 
							int dim, 
							int numParticles, 
							double * pos_vec_array, 
							double * vel_vec_array, 
							double a_1, double a_2, 
							double * P_b, double * P_g, 
							double *next_pos_vec_array)
{
	// Initialize the particles

	// Using the current position of the particles (from pos_vec_array), compute the score at each particle

	// Using the current position of the particles (from pos_vec_array), Update the Personal best for each particle

	// Of all the particles, do a maximum reduction on global data to find the global max

	// Randomly generate the two random vectors [0,1]

	// Move the particles and update the positions

	// Compute the convergence metric

	// If done, then exit

	// Else, repeat up to max num times
	

}

}
