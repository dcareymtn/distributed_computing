#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <time.h>

#include "gpu_algo.h"

namespace gpu
{

__global__ void print_cuda(char *a, int N)
{
    char p[11]="Hello CUDA";
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    
	if(idx < N) 
    {
        a[idx]=p[idx];
    }
}

__global__ void gpu_count_occurrences_global( double *dM, int size, int start_index, int stop_index, int *d_counter )
{
	int tid = blockIdx.x * blockDim.x + threadIdx.x;
	
	if (tid > size) return;

	int bin = dM[tid] - start_index;

	atomicAdd( &d_counter[bin], 1);

}

__global__ void gpu_count_occurrences_shared( double * pM, int size, int start_index, int stop_index, int *d_counter )
{

	extern __shared__ int s_counter[];

	int tid 	= blockIdx.x * blockDim.x + threadIdx.x;
	int N = stop_index - start_index + 1;

	if (tid < (stop_index - start_index + 1))
	{	
		s_counter[tid] = 0;
		d_counter[tid] = 0;
	}

	__syncthreads();
	
	if (tid < size)
	{
		int sbin = pM[tid] - start_index;
		atomicAdd( &s_counter[sbin], 1);
	}

	__syncthreads();

	if ((threadIdx.x < N) && (tid < size))
	{
		atomicAdd( &d_counter[threadIdx.x], s_counter[threadIdx.x] );
	}

}

__global__ void gpu_rms_filter_global( double * _d_M, int filtNRows, int filtNCols, double * _d_MResult )
{
	int subMatIdx = blockIdx.x;
	
	int blockSize	 = blockDim.x * blockDim.y;

	int row = threadIdx.y;
	int col = threadIdx.x;

	double this_result;

	int start_row 	= max(row - (filtNRows - 1), 0);
	int stop_row 	= min(row + (filtNRows - 1), blockDim.y);
	
	int start_col 	= max(col - (filtNCols - 1), 0);
	int stop_col 	= min(col + (filtNCols - 1), blockDim.x);

	double temp(0);

	for (int iRow = start_row; iRow < stop_row; iRow++)
	{
		for (int iCol = start_col; iCol < stop_col; iCol++)
		{
			temp 		= *(_d_M + subMatIdx * blockSize + iRow * blockDim.y + iCol );
			this_result += temp*temp;


		}	
	}
	
	*(_d_MResult + subMatIdx * blockSize + row * blockDim.y + col ) = this_result;
}

void cuda_init()
{
	double *dM;
	hipMalloc( (void **) &dM, 10*sizeof(double));
	hipFree(dM);
}

void count_occurrences( double *h_M, int nRows, int nCols, int start_count, int stop_count )
{
	
	// Copy the matrix data to the gpu
	double *d_M;
	int *d_counter;
		
	int nbins 	= stop_count - start_count + 1;

	size_t counter_size = nbins * sizeof( int );
	int *h_counter = (int *) malloc( counter_size );

	for (int ii = 0; ii < nbins; ii++)
	{
		h_counter[ii] = 0;
	}

	int N = nRows * nCols;

	size_t size = N * sizeof( double );

	hipMalloc((void **)&d_M, size);
	hipMalloc((void **)&d_counter, counter_size);

	hipMemcpy( d_M, h_M, size, hipMemcpyHostToDevice);
	
	int factor = 4;

	int blockSize = (factor * nCols < 1000 ? factor * nCols : 1000);
	int nBlock = N / blockSize + (N%blockSize == 0 ? 0 : 1);
	
	gpu_count_occurrences_shared<<< nBlock, blockSize, nbins*sizeof(int)  >>>(d_M, size, start_count, stop_count, d_counter );
	//gpu_count_occurrences_global<<< nBlock, blockSize  >>>(d_M, size, start_count, stop_count, d_counter );

	hipMemcpy( h_counter, d_counter, counter_size, hipMemcpyDeviceToHost);

	for (int ii = start_count; ii <= stop_count; ii++)
	{
		printf("%4d | ", ii);
	}
	printf("\n");
	for (int ii = 0; ii < nbins; ii++)
	{
		printf("%3d |  ", h_counter[ii]);
	}
	printf("\n");

	free(h_counter);
	hipFree(d_M);
	hipFree(d_counter);
}	

void rms_filter( double *hBlockMResult, double *_h_BlockM, int nRowBreak, int subMatNumRows, int subMatNumCols, int nFiltRows, int nFiltCols)
{
	double *_d_BlockM, *_d_BlockMResult;

	size_t __blockSize = nRowBreak * subMatNumRows * subMatNumCols * sizeof(double);

	hipMalloc((void **)&_d_BlockM, __blockSize );
	hipMalloc((void **)&_d_BlockMResult, __blockSize );

	hipMemcpy( _d_BlockM, _h_BlockM, __blockSize, hipMemcpyHostToDevice );

	dim3 threadsPerBlock(subMatNumRows, subMatNumCols );

	gpu_rms_filter_global<<< nRowBreak, threadsPerBlock >>>( _d_BlockM, nFiltRows, nFiltCols, _d_BlockMResult );

	hipMemcpy( hBlockMResult, _d_BlockMResult, __blockSize, hipMemcpyDeviceToHost );
	
	hipFree(_d_BlockM);
	hipFree(_d_BlockMResult);	
}

void hello_cuda(void)
{
    char *a_h, *a_d; // _h for the host and _d for the device based pointers
    const int N = 11;
    size_t size = N * sizeof(char);

    a_h = (char *) malloc(size); // allocating the array on the host
    hipMalloc((void **) &a_d, size); // allocating the array on the device
    
    // initialize the host array
    for (int i = 0; i < N; i++)
    {
        a_h[i] = 0;
    }

    // Copy the array on the host to the device
    hipMemcpy(a_d, a_h, size, hipMemcpyHostToDevice);

    int blocksize = 4;
    int nblock = N/blocksize + (N%blocksize == 0 ? 0 : 1);
	nblock = 100;

    print_cuda <<< nblock, blocksize >>>(a_d, N); // Run the kernel on the device

	hipError_t err = hipGetLastError();
	if (err != hipSuccess)
		printf("Error: %s\n", hipGetErrorString(err));
    // copy from the device memory to the host memory
    hipMemcpy(a_h, a_d, sizeof(char) * N, hipMemcpyDeviceToHost);


	// print the array on the host
    for (int i = 0; i < N; i++)
    {
        printf("%c", a_h[i]);
    }
	printf("\n");
    free(a_h);
	hipFree(a_d);
}

}
