#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

#include "gpu_algo.h"

namespace gpu
{

__global__ void print_cuda(char *a, int N)
{
    char p[11]="Hello CUDA";
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
	//printf("Hello\n");
    if(idx < N) 
    {
        a[idx]=p[idx];
    }
}

__global__ void gpu_count_occurrences( double * pM, int size, int start_index, int *d_counter )
{

	int tid 	= blockIdx.x * blockDim.x + threadIdx.x;
	
	// Check that the thread ID is in the matrix
	if (tid >= size) 
	{ 
		return;
	}

	int bin = pM[tid] - start_index;
	
	atomicAdd( &d_counter[bin], 1 );
}

void count_occurrences( double *h_M, int nRows, int nCols, int start_count, int stop_count )
{
	
	// Copy the matrix data to the gpu
	double *d_M;
	int *d_counter;
	
	int nbins 	= stop_count - start_count + 1;

	size_t counter_size = nbins * sizeof( int );
	int *h_counter = (int *) malloc( counter_size );

	for (int ii = 0; ii < nbins; ii++)
	{
		h_counter[ii] = 0;
	}

	int N = nRows * nCols;

	size_t size = N * sizeof( double );
	
	hipMalloc((void **)&d_M, size);
	hipMalloc((void **)&d_counter, counter_size);
	
	hipMemcpy( d_M, h_M, size, hipMemcpyHostToDevice);
	
	int blockSize = nCols;
	int nBlock = N / blockSize + (N%blockSize == 0 ? 0 : 1);

	gpu_count_occurrences<<< nBlock, blockSize >>>(d_M, size, start_count, d_counter);

	hipMemcpy( h_counter, d_counter, counter_size, hipMemcpyDeviceToHost);

	for (int ii = start_count; ii <= stop_count; ii++)
	{
		printf("%d ", ii);
	}
	printf("\n");
	for (int ii = 0; ii < nbins; ii++)
	{
		printf("%d ", h_counter[ii]);
	}
	printf("\n");

	free(h_counter);
	hipFree(d_M);
	hipFree(d_counter);
}	

void hello_cuda(void)
{
    char *a_h, *a_d; // _h for the host and _d for the device based pointers
    const int N = 11;
    size_t size = N * sizeof(char);

    a_h = (char *) malloc(size); // allocating the array on the host
    hipMalloc((void **) &a_d, size); // allocating the array on the device
    
    // initialize the host array
    for (int i = 0; i < N; i++)
    {
        a_h[i] = 0;
    }

    // Copy the array on the host to the device
    hipMemcpy(a_d, a_h, size, hipMemcpyHostToDevice);

    int blocksize = 4;
    int nblock = N/blocksize + (N%blocksize == 0 ? 0 : 1);
	nblock = 100;

    print_cuda <<< nblock, blocksize >>>(a_d, N); // Run the kernel on the device

	hipError_t err = hipGetLastError();
	if (err != hipSuccess)
		printf("Error: %s\n", hipGetErrorString(err));
    // copy from the device memory to the host memory
    hipMemcpy(a_h, a_d, sizeof(char) * N, hipMemcpyDeviceToHost);


	// print the array on the host
    for (int i = 0; i < N; i++)
    {
        printf("%c", a_h[i]);
    }
	printf("\n");
    free(a_h);
	hipFree(a_d);
}

}
